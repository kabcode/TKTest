#include "hip/hip_runtime.h"
#include "CudaHost.hcu"
#include "iostream"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

CudaHost::CudaHost()
{
    printf("CudaHost::CudaHost()\n");
    InstantiateCudaDeviceClass<<<1,1>>>(cudadevice_d);
    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());
}

CudaHost::~CudaHost()
{
    DeleteCudaDeviceClass << <1, 1 >> > (cudadevice_d);
    cudadevice_d = nullptr;
}

void CudaHost::Run()
{
    printf("CudaHost::Run()\n");
    RunDeviceKernels <<<1, 1 >>> (*cudadevice_d);
    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());
}

__global__ void InstantiateCudaDeviceClass(CudaDevice* d_ptr)
{
    printf("__global__ CudaHost::InstantiateCudaDeviceClass()\n");
    d_ptr = new CudaDevice();
    gpuErrchk(hipPeekAtLastError());
}

__global__ void RunDeviceKernels(CudaDevice& d_ptr)
{
    printf("__global__ CudaHost::RunDeviceKernels()\n");
    d_ptr.Run();
    gpuErrchk(hipPeekAtLastError());
}

__global__ void DeleteCudaDeviceClass(CudaDevice* d_ptr)
{
    printf("__global__ CudaHost::DeleteCudaDeviceClass()\n");
    delete d_ptr;
    gpuErrchk(hipPeekAtLastError());
}
