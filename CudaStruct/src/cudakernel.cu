#include "hip/hip_runtime.h"
#include "cudakernel.hcu"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""
#include "iostream"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__constant__ unsigned Size[4];
__constant__ float Spacing[4];

template<unsigned int TImageDimension>
__global__ void kernel(float* indata, float* outdata)
{
    const auto i = blockIdx.x * blockDim.x + threadIdx.x;
    const auto j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= Size[0] || j >= Size[1])
        return;

    outdata[j + i * Size[1]] = indata[j + i * Size[1]] * (Size[0] + Spacing[1]);
    printf("[%u,%u] -> %.2f -> %.2f\n", i, j, indata[j + i * Size[1]], outdata[j + i * Size[1]]);
}

template<unsigned Dimension>
void
CUDA_copyToConstant(unsigned* size, float* spacing)
{
    hipMemcpyToSymbol(HIP_SYMBOL(Size), size, 4 * sizeof(unsigned int));
    hipMemcpyToSymbol(HIP_SYMBOL(Spacing), spacing, 4 * sizeof(float));

    unsigned int checkSize[4];
    float checkSpacing[4];

    hipMemcpyFromSymbol(checkSize, HIP_SYMBOL(Size), 4 * sizeof(unsigned int), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(checkSpacing, HIP_SYMBOL(Spacing), 4 * sizeof(float), 0, hipMemcpyDeviceToHost);
}

template<unsigned TImageDimension>
void
CUDA_testkernel(float* indata, float* outdata)
{
   
    float* d_data;
    float* d_indata;

    unsigned int checkSize[4];
    hipMemcpyFromSymbol(checkSize, HIP_SYMBOL(Size), 4 * sizeof(unsigned int), 0, hipMemcpyDeviceToHost);

    long int outputmemory = 1;
    for (auto i = 0; i < TImageDimension; ++i)
    {
        outputmemory *= checkSize[i];
    }
    gpuErrchk(hipMalloc((void**)&d_data, outputmemory * sizeof(float)));
    gpuErrchk(hipMemset(d_data, 0, outputmemory * sizeof(float)));

    gpuErrchk(hipMalloc((void**)&d_indata, outputmemory * sizeof(float)));
    gpuErrchk(hipMemcpy(d_indata, indata, outputmemory * sizeof(float), hipMemcpyHostToDevice));

    auto dimBlock = dim3(16, 16);
    auto blocksInX = std::ceil(checkSize[0] / dimBlock.x);
    auto blocksInY = std::ceil(checkSize[1] / dimBlock.y);
    blocksInX = blocksInX < 1 ? 1 : blocksInX;
    blocksInY = blocksInY < 1 ? 1 : blocksInY;
    auto dimGrid = dim3(blocksInX, blocksInY);

    kernel<TImageDimension> << < dimGrid, dimBlock >> > (d_indata, d_data);
    gpuErrchk(hipDeviceSynchronize());

    hipMemcpy(outdata, d_data, outputmemory * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
   
    hipFree(d_data);
    hipFree(d_indata);
};


template void CUDA_testkernel<2>(float*, float*);
template void CUDA_testkernel<3>(float*, float*);

template void CUDA_copyToConstant<2>(unsigned*, float*);
template void CUDA_copyToConstant<3>(unsigned*, float*);