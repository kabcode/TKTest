#include "hip/hip_runtime.h"
#include <VectorKernel.hcu>

#include <hip/hip_runtime.h>
#include "rtkCudaUtilities.hcu"
#include <hip/hip_runtime_api.h>
#include <>
#include <vector_functions.hpp>
#include <hip/hip_runtime.h>

__constant__ int3 c_size;

template<unsigned veclen>
__global__ void useMultipleTextures(hipTextureObject_t* tex, float* pout, int* pts)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int k = blockIdx.z * blockDim.z + threadIdx.z;

	float sample[veclen];
	for (unsigned int c = 0; c < veclen; c++)
		sample[c] = tex3D<float>(tex[c], i + 0.5, j + 0.5, k + 0.5);

	auto magn = 0.f;
	for (unsigned int c = 0; c < veclen; c++)
		magn += powf(sample[c], 2);

	pout[i + c_size.x * (j + k * c_size.y)] = sqrtf(magn);
}

__global__ void useSingleTexture(hipTextureObject_t tex, float* pout)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int k = blockIdx.z * blockDim.z + threadIdx.z;

	float4 sample = tex3D<float4>(tex, i + 0.5, j + 0.5, k + 0.5);

	pout[i + c_size.x * (j + k * c_size.y)] = sqrtf(powf(sample.x,2)+ powf(sample.y, 2)+ powf(sample.z, 2));
}

__global__ void useNoTexture(float* pin, float* pout, int len)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int k = blockIdx.z * blockDim.z + threadIdx.z;

	auto a = pin[0 + len * (i + c_size.x * (j + k * c_size.y))];
	auto b = pin[1 + len * (i + c_size.x * (j + k * c_size.y))];
	auto c = pin[2 + len * (i + c_size.x * (j + k * c_size.y))];

	pout[i + c_size.x * (j + k * c_size.y)] = sqrtf(powf(a, 2) + powf(b, 2) + powf(c, 2));
	
}


void
CUDA_useMultipleTextures(float* dev_in, float* pout, int* size, int length, int* pts)
{
	
	hipMemcpyToSymbol(HIP_SYMBOL(c_size), size, sizeof(int3));

	int bytes = 3 * 10000 * sizeof(int);
	int* dev_pts;
	hipMalloc(&dev_pts, bytes);
	hipMemcpy(dev_pts, pts, bytes, hipMemcpyHostToDevice);

	hipTextureObject_t* tex_vol = new hipTextureObject_t[length];
	hipArray** volCompArrays = new hipArray*[length];

	prepareTextureObject(size, dev_in, volCompArrays, length, tex_vol, false);
	hipTextureObject_t* dev_tex_vol;
	hipMalloc(&dev_tex_vol, length * sizeof(hipTextureObject_t));
	hipMemcpy(dev_tex_vol, tex_vol, length * sizeof(hipTextureObject_t), hipMemcpyHostToDevice);

	dim3 dimBlock = dim3(256, 1, 1);

	int blocksInX = iDivUp(size[0], dimBlock.x);
	int blocksInY = iDivUp(size[1], dimBlock.y);
	int blocksInZ = iDivUp(size[2], dimBlock.z);

	dim3 dimGrid = dim3(blocksInX, blocksInY, blocksInZ);
	useMultipleTextures<3> <<<dimGrid, dimBlock >>> (dev_tex_vol, pout, dev_pts);

	CUDA_CHECK_ERROR;
}

void
CUDA_useSingleTexture(float* dev_in, float* pout, int* size, int length)
{

	hipMemcpyToSymbol(HIP_SYMBOL(c_size), size, sizeof(int3));
	
	// insert a new 0 at every fourth position
	auto ar = malloc(size[0] * size[1] * size[2] * sizeof(float4));
	for (auto k = 0; k < size[2]; ++k)
	{
		for (auto j = 0; j < size[1]; ++j)
		{
			for (auto i = 0; i < size[0]; ++i)
			{
				
			}
		}
	}

	// Allocate CUDA array in device memory
	auto channelDesc = hipCreateChannelDesc<float4>();
	auto volExtent = make_hipExtent(size[0], size[1], size[2]);
	hipArray* volArray = nullptr;
	hipMalloc3DArray((hipArray**)& volArray, &channelDesc, volExtent);
	hipMemcpy3DParms CopyParams = { 0 };
	CopyParams.srcPtr = make_hipPitchedPtr((void*)dev_in, size[0] * sizeof(float4), size[0], size[1]);
	CopyParams.dstArray = volArray;
	CopyParams.extent = volExtent;
	CopyParams.kind = hipMemcpyHostToDevice;
	hipMemcpy3D(&CopyParams);
	CUDA_CHECK_ERROR;

	// Specify texture
	struct hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = volArray;

	// Specify texture object parameters
	struct hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.addressMode[0] = hipAddressModeClamp;
	texDesc.addressMode[1] = hipAddressModeClamp;
	texDesc.addressMode[2] = hipAddressModeClamp;
	texDesc.filterMode = hipFilterModeLinear;
	texDesc.readMode = hipReadModeElementType;
	texDesc.normalizedCoords = 0;

	// Create texture object
	hipTextureObject_t texObj = 0;
	hipCreateTextureObject(&texObj, &resDesc, &texDesc, nullptr);
	

	dim3 dimBlock = dim3(8, 8, 8);

	int blocksInX = iDivUp(size[0], dimBlock.x);
	int blocksInY = iDivUp(size[1], dimBlock.y);
	int blocksInZ = iDivUp(size[2], dimBlock.z);

	dim3 dimGrid = dim3(blocksInX, blocksInY, blocksInZ);
	useSingleTexture<<<dimGrid, dimBlock >>> (texObj, pout);

	CUDA_CHECK_ERROR;
}

void
CUDA_useNoTexture(float * pin, float* pout, int* size, int length)
{
	hipMemcpyToSymbol(HIP_SYMBOL(c_size), size, sizeof(int3));
	
	dim3 dimBlock = dim3(8, 8, 8);

	int blocksInX = iDivUp(size[0], dimBlock.x);
	int blocksInY = iDivUp(size[1], dimBlock.y);
	int blocksInZ = iDivUp(size[2], dimBlock.z);

	dim3 dimGrid = dim3(blocksInX, blocksInY, blocksInZ);
	useNoTexture <<<dimGrid, dimBlock >>> (pin, pout, length);

	CUDA_CHECK_ERROR;
}

__host__  void prepareTextureObject(int size[3],
                                    float *dev_ptr,
                                    hipArray **&componentArrays,
                                    unsigned int nComponents,
                                    hipTextureObject_t *tex,
                                    bool isProjections)
{
	// Create CUBLAS context
	hipblasHandle_t  handle;
	hipblasCreate(&handle);

	// create texture object
	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;

	hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.readMode = hipReadModeElementType;
	for (unsigned int component = 0; component < nComponents; component++)
	{
		if (isProjections)
			texDesc.addressMode[component] = hipAddressModeBorder;
		else
			texDesc.addressMode[component] = hipAddressModeClamp;
	}
	texDesc.filterMode = hipFilterModeLinear;

	static hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	hipExtent volExtent = make_hipExtent(size[0], size[1], size[2]);

	// Allocate an intermediate memory space to extract the components of the input volume
	float *singleComponent;
	int numel = size[0] * size[1] * size[2];
	hipMalloc(&singleComponent, numel * sizeof(float));
	float one = 1.0;

	// Copy image data to arrays. The tricky part is the make_hipPitchedPtr.
	// The best way to understand it is to read
	// http://stackoverflow.com/questions/16119943/how-and-when-should-i-use-pitched-pointer-with-the-cuda-api
	for (unsigned int component = 0; component < nComponents; component++)
	{
		// Reset the intermediate memory
		hipMemset((void *)singleComponent, 0, numel * sizeof(float));

		// Fill it with the current component
		float * pComponent = dev_ptr + component;
		hipblasSaxpy(handle, numel, &one, pComponent, nComponents, singleComponent, 1);

		// Allocate the hipArray. Projections use layered arrays, volumes use default 3D arrays
		if (isProjections)
			hipMalloc3DArray((hipArray**)& componentArrays[component], &channelDesc, volExtent, hipArrayLayered);
		else
			hipMalloc3DArray((hipArray**)& componentArrays[component], &channelDesc, volExtent);

		// Fill it with the current singleComponent
		hipMemcpy3DParms CopyParams = { 0 };
		CopyParams.srcPtr = make_hipPitchedPtr(singleComponent, size[0] * sizeof(float), size[0], size[1]);
		CopyParams.dstArray = (hipArray*)componentArrays[component];
		CopyParams.extent = volExtent;
		CopyParams.kind = hipMemcpyDeviceToDevice;
		hipMemcpy3D(&CopyParams);
		CUDA_CHECK_ERROR;

		// Fill in the texture object with all this information
		resDesc.res.array.array = componentArrays[component];
		hipCreateTextureObject(&tex[component], &resDesc, &texDesc, NULL);
		CUDA_CHECK_ERROR;
	}

	// Intermediate memory is no longer needed
	hipFree(singleComponent);

	// Destroy CUBLAS context
	hipblasDestroy(handle);
}

